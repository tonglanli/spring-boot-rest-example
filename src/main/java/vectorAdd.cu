
#include <hip/hip_runtime.h>
extern "C"
__global__ void add(int n, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = x[i] + y[i];
}